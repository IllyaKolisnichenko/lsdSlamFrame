#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

hipDeviceProp_t deviceProps;

bool hipInit()
{
    bool    res     = true;
    int     device  = 0;

    hipError_t err = hipGetDevice( &device );

    if( err != hipSuccess )
    {
        printf("%s\n", hipGetErrorString(err));

        res = false;

        return res;
    }

    printf("  Device Count: %d\n", device+1 );

    for(int i = 0; i <= device; i++)
    {
        hipGetDeviceProperties(&deviceProps, device);

        printf("  Device Number: %d\n", i);
        printf("  Device name: %s\n", deviceProps.name);

        printf("  Memory Clock Rate (KHz): %d\n", deviceProps.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n", deviceProps.memoryBusWidth );

        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*deviceProps.memoryClockRate*(deviceProps.memoryBusWidth/8)/1.0e6);


        printf("  Max Threads per Block: %d\n", deviceProps.maxThreadsPerBlock );
    }

    // Select CUDA device
    hipSetDevice(0);

    return res;
}

//******************************************************************************************************

// !!! Exemple DO NOT TOCH !!!!!
//__global__ void findMean(unsigned int dataForBlock, float *inputData, float *results)
//{
//    int index = blockIdx.x * blockDim.x + threadIdx.x;
//    float result = 0;

//    for (int i = 0; i < dataForBlock; i++)
//    {
//        result += inputData[index * dataForBlock + i];
//    }

//    result /= dataForBlock;
//    results[index] = result;
//}

//void processWithGPU(float *blocks, float *results, unsigned int blockSize, unsigned int blocksCount)
//{
//    unsigned int realDataCount = blockSize * blocksCount;

//    hipSetDevice(0);

//    float   *deviceInputData,
//            *deviceResults;
//    hipMalloc( (void**)&deviceInputData,  realDataCount * sizeof(float)   );
//    hipMalloc( (void**)&deviceResults,    realDataCount * sizeof(float)   );

//    hipMemcpy( deviceInputData, blocks, realDataCount * sizeof(float), hipMemcpyHostToDevice );

//    findMean<<<1, blocksCount>>>( blockSize, deviceInputData, deviceResults );

//    hipMemcpy( (void*)results, deviceResults, blocksCount * sizeof(float), hipMemcpyDeviceToHost );

//    hipFree(deviceInputData);
//    hipFree(deviceResults  );
//}
//********************************************************************************************************
// Build Image
__global__ void cuBuildImageKernel( float* source, float* dest )
{
    int x       = blockIdx.x;
    int y       = blockIdx.y;

    int offset  = x + y * gridDim.x;

    float* s = source + ( y * (gridDim.x * 2 ) * 2 + x * 2);

	float t_[4], t_sume, t_result;
	// Save temp values
	t_[0] = s[0];
	t_[1] = s[1];
	t_[2] = s[	gridDim.x * 2];
	t_[3] = s[1+gridDim.x * 2];

	// Calculate sum
	t_sume = t_[0] + t_[1] + t_[2] + t_[3];

	// Calculate result
	t_result = t_sume * 0.25;


    dest[offset] = (	s[ 0 ] +
              			s[ 1 ] +
                     	s[ gridDim.x * 2 ]   +
                     	s[ gridDim.x * 2 + 1]   ) * 0.25f;
}

void cuBuildImage(  const float* source,    int sourceWidth,    int sourceHeight,
                    const float* dest,      int destWidth,      int destHeight      )
{

    int sourceBuffLength    = sourceWidth   * sourceHeight;
    int destBuffLength      = destWidth     * destHeight;

    // Reserving memory on GPU
    float   *sourceBuff,
            *destBuff;

    hipMalloc( (void**)&sourceBuff,    sourceBuffLength  * sizeof(float)   );
    hipMalloc( (void**)&destBuff,      destBuffLength    * sizeof(float)   );

    // Copy input buffer
    hipMemcpy( sourceBuff, source,     sourceBuffLength  * sizeof(float), hipMemcpyHostToDevice );

    dim3 grid( destWidth, destHeight );
    cuBuildImageKernel<<<grid, 1>>>( sourceBuff,  destBuff );

    hipMemcpy( (void*)dest, destBuff,  destBuffLength * sizeof(float), hipMemcpyDeviceToHost );

    hipFree( sourceBuff );
    hipFree( destBuff   );
}

//********************************************************************************************************
// Build Gradient
//__global__ void cuBuildGradientsKernel()
//{

//}

//void cuBuildGradients(const float* )
//{
//    const float*    img_pt      = data.image[level]     + width;
//    const float*    img_pt_max  = data.image[level]     + width * (height-1);
//    float*          gradxyii_pt = data.gradients[level] + width;

//    // in each iteration i need -1,0,p1,mw,pw
//    float val_m1 = *(img_pt-1);
//    float val_00 = * img_pt;
//    float val_p1;

//    for(; img_pt < img_pt_max; img_pt++, gradxyii_pt++)
//    {
//        val_p1 = *(img_pt+1);

//        *( (float*)gradxyii_pt +0)  = 0.5f*(val_p1 - val_m1);
//        *(((float*)gradxyii_pt)+1)  = 0.5f*(*(img_pt+width) - *(img_pt-width));
//        *(((float*)gradxyii_pt)+2)  = val_00;

//        val_m1 = val_00;
//        val_00 = val_p1;
//    }
//}

//********************************************************************************************************
// Build MaxGradient
//__global__ void cuBuildMaxGradientsKernel()
//{

//}

//void buildMaxGradients(int level)
//{

//    float* maxGradTemp = FrameMemory::getInstance().getFloatBuffer(width * height);

//    // 1. write abs gradients in real data.
//    Eigen::Vector4f* gradxyii_pt = data.gradients[level] + width;

//    float* maxgrad_pt       = data.maxGradients[level] + width;
//    float* maxgrad_pt_max   = data.maxGradients[level] + width*(height-1);

//    for(; maxgrad_pt < maxgrad_pt_max; maxgrad_pt++, gradxyii_pt++ )
//    {
//        float dx = *(  (float*)gradxyii_pt);
//        float dy = *(1+(float*)gradxyii_pt);
//        *maxgrad_pt = sqrtf(dx*dx + dy*dy);
//    }

//    // 2. smear up/down direction into temp buffer
//    maxgrad_pt      = data.maxGradients[level] + width+1;
//    maxgrad_pt_max  = data.maxGradients[level] + width*(height-1)-1;

//    float* maxgrad_t_pt = maxGradTemp + width+1;
//    for(;maxgrad_pt<maxgrad_pt_max; maxgrad_pt++, maxgrad_t_pt++ )
//    {
//        float g1 = maxgrad_pt[-width];
//        float g2 = maxgrad_pt[0];

//        if(g1 < g2)
//            g1 = g2;

//        float g3 = maxgrad_pt[width];

//        if(g1 < g3)
//            *maxgrad_t_pt = g3;
//        else
//            *maxgrad_t_pt = g1;
//    }

//    float numMappablePixels = 0;

//    // 2. smear left/right direction into real data
//    maxgrad_pt      = data.maxGradients[level] + width+1;
//    maxgrad_pt_max  = data.maxGradients[level] + width*(height-1)-1;
//    maxgrad_t_pt    = maxGradTemp + width+1;
//    for(;maxgrad_pt<maxgrad_pt_max; maxgrad_pt++, maxgrad_t_pt++ )
//    {
//        float g1 = maxgrad_t_pt[-1];
//        float g2 = maxgrad_t_pt[0];

//        if(g1 < g2)
//            g1 = g2;

//        float g3 = maxgrad_t_pt[1];
//        if(g1 < g3)
//        {
//            *maxgrad_pt = g3;
//            if(g3 >= MIN_ABS_GRAD_CREATE)
//                numMappablePixels++;
//        }
//        else
//        {
//            *maxgrad_pt = g1;
//            if(g1 >= MIN_ABS_GRAD_CREATE)
//                numMappablePixels++;
//        }
//    }

//    if(level==0)
//        this->numMappablePixels = numMappablePixels;

//    FrameMemory::getInstance().returnBuffer(maxGradTemp);
//}
//********************************************************************************************************
//__global__ void cuBuildIDepthAndIDepthVarKernel()
//{

//}

//// Build IDepth And IDepth Var
//void buildIDepthAndIDepthVar( int level )
//{
//    int sw = data.width[level - 1];

//    const float* idepthSource       = data.idepth   [level - 1];
//    const float* idepthVarSource    = data.idepthVar[level - 1];

//    float* idepthDest       = data.idepth   [level];
//    float* idepthVarDest    = data.idepthVar[level];

//    for( int y = 0; y < height; y++ )
//    {
//        for( int x = 0; x < width; x++ )
//        {
//            int idx     = 2 * ( x + y * sw  );
//            int idxDest = ( x + y * width   );

//            float idepthSumsSum = 0;
//            float ivarSumsSum   = 0;
//            int   num           = 0;

//            // build sums
//            float ivar;
//            float var = idepthVarSource[idx];
//            if( var > 0 )
//            {
//                ivar             = 1.0f / var;
//                ivarSumsSum     += ivar;
//                idepthSumsSum   += ivar * idepthSource[idx];
//                num++;
//            }

//            var = idepthVarSource[ idx + 1 ];
//            if( var > 0 )
//            {
//                ivar             = 1.0f / var;
//                ivarSumsSum     += ivar;
//                idepthSumsSum   += ivar * idepthSource[ idx + 1 ];
//                num++;
//            }

//            var = idepthVarSource[ idx + sw ];
//            if( var > 0 )
//            {
//                ivar             = 1.0f / var;
//                ivarSumsSum     += ivar;
//                idepthSumsSum   += ivar * idepthSource[ idx + sw ];
//                num++;
//            }

//            var = idepthVarSource[ idx + sw + 1 ];
//            if( var > 0 )
//            {
//                ivar             = 1.0f / var;
//                ivarSumsSum     += ivar;
//                idepthSumsSum   += ivar * idepthSource[ idx + sw + 1 ];
//                num++;
//            }

//            if(num > 0)
//            {
//                float depth = ivarSumsSum / idepthSumsSum;

//                idepthDest   [ idxDest ] = 1.0f / depth;
//                idepthVarDest[ idxDest ] = num  / ivarSumsSum;
//            }
//            else
//            {
//                idepthDest   [ idxDest ] = -1;
//                idepthVarDest[ idxDest ] = -1;
//            }
//        }
//    }
//}
